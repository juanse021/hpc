#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <malloc.h>
#include <opencv2/opencv.hpp>

using namespace cv;

__global__ void grayImageDevice(unsigned char *image_begin, int width, int height, unsigned char *image_end) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < height) && (col < width)) {
        image_end[row*width+col] = image_begin[(row*width+col) * 3 + 2] * 0.3 + image_begin[(row*width+col) * 3 + 1] * 0.59 + image_begin[(row*width+col) * 3] * 0.11;
    }
}

int main(int argc, char **argv) {
    char *imageName = argv[1];
    Mat image = imread(imageName, 1);

    if (!image.data) {
        printf("Could not open or find the image \n");
        return -1;
    }

    Size s = image.size();
    int width = s.width;
    int height = s.height;

    hipError_t error = hipSuccess;
    int size = width * height * sizeof(unsigned char) * image.channels();
    int sizeGray = width * height * sizeof(unsigned char);


    unsigned char *h_imageA, *h_imageB, *d_imageA, *d_imageB;

    // h_imageA = (unsigned char*)malloc(size);
    error = hipMalloc((void**)&d_imageA, size);
    if (error != hipSuccess) {
        printf("Error.... d_imageA \n");
        return -1;
    }

    h_imageA = image.data;

    error = hipMemcpy(d_imageA, h_imageA, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error... h_imageA a d_imageA \n");
        return -1;
    }

    // Separar memoria de imagen gris en host
    h_imageB = (unsigned char*)malloc(sizeGray);
    error = hipMalloc((void**)&d_imageB, sizeGray);
    if (error != hipSuccess) {
        printf("Error.... d_imageB \n");
        return -1;
    }

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(ceil(height/32.0), ceil(width/32.0), 1);

    grayImageDevice<<<dimGrid, dimBlock>>>(d_imageA, width, height, d_imageB);
    hipDeviceSynchronize();

    error = hipMemcpy(h_imageB, d_imageB, sizeGray, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Error... d_imageB a h_imageB \n");
        return -1;
    }

    //grayImageHost(h_imageA, width, height, h_imageB);

    Mat imageGray;
    imageGray.create(height, width, CV_8UC1);
    imageGray.data = h_imageB;

    imwrite("ferrari_gray.jpg", imageGray);


    free(h_imageB);
    hipFree(d_imageA); hipFree(d_imageB);

    return 0;
}
