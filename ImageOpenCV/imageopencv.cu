#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cv.h>
#include <highgui.h>
#include <stdio.h>
#include <time.h>
#include <fstream>

__global__ void grayImage(unsigned char *image_begin, int width, int height, unsigned char *image_end) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < height) && (col < width)) {
        image_end[row*width+col] = image_end[(row*width+col) * 3] * 0.3 + \
        image_end[(row*width+col) * 3] * 0.59 + \
        image_end[(row*width+col) * 3] * 0.11;
    }
}

int main(int argc, char **argv) {
    char *imageName = argv[1];
    Mat image = imread(imageName, 1);
    int width = image.size().width();
    int height = image.size().height();

    if (!image.data) {
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

    hipError_t error = hipSuccess;
    int size = width * height * 3 * sizeof(unsigned char*);
    unsigned char *h_imageA, *h_imageB, *d_imageA, *d_imageB;

    // Separar memoria de imagen color en host
    h_imageA = (unsigned char*)malloc(size));
    error = hipMalloc((void**)&d_imageA, sizeof(unsigned char*) * width * height);
    if (error != hipSuccess) {
        printf("Error.... d_imageA \n");
        return -1;
    }
    // Separar memoria de imagen gris en host
    h_imageB = (unsigned char*)malloc(size));
    error = hipMalloc((void**)&d_imageB, sizeof(unsigned char*) * width * height);
    if (error != hipSuccess) {
        printf("Error.... d_imageB \n");
        return -1;
    }

    h_imageA = image.data;

    error = hipMemcpy(d_imageA, h_imageA, size,hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error... h_imageA a d_imageA \n");
        return -1;
    }

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(ceil(width/float(32)), ceil(height/float(32)), 1);
    grayImage<<<dimGrid, dimBlock>>>(d_imageA, width, height, d_imageB);
    hipDeviceSynchronize();

    error = hipMemcpy(h_imageB, d_imageB, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Error... d_imageB a h_imageB \n");
        return -1;
    }

    Mat imageGray;
    imageGray.create(width, heigth, CV_8UC1);
    imageGray.data = d_imageB;



    return 0;
}
